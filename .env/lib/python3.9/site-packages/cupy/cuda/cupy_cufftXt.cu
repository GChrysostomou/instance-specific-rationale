#include "cupy_cufftXt.h"


// this must define d_loadCallbackPtr
${dev_load_callback_ker}

// this must define d_storeCallbackPtr
${dev_store_callback_ker}

hipfftResult set_callback(hipfftHandle plan, hipfftXtCallbackType type, bool cb_load, void** callerInfo) {
    if (cb_load) {
        switch (type) {
            #ifdef HAS_LOAD_CALLBACK
            case HIPFFT_CB_LD_COMPLEX: {
                hipfftCallbackLoadC h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_LD_COMPLEX_DOUBLE: {
                hipfftCallbackLoadZ h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_LD_REAL: {
                hipfftCallbackLoadR h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_LD_REAL_DOUBLE: {
                hipfftCallbackLoadD h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_loadCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            #endif  // HAS_LOAD_CALLBACK
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    } else {
        switch (type) {
            #ifdef HAS_STORE_CALLBACK
            case HIPFFT_CB_ST_COMPLEX: {
                hipfftCallbackStoreC h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_ST_COMPLEX_DOUBLE: {
                hipfftCallbackStoreZ h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_ST_REAL: {
                hipfftCallbackStoreR h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            case HIPFFT_CB_ST_REAL_DOUBLE: {
                hipfftCallbackStoreD h_ptr;
                hipMemcpyFromSymbol(&h_ptr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_ptr));
                return hipfftXtSetCallback(plan, (void**)&h_ptr, type, callerInfo);
            }
            #endif  // HAS_STORE_CALLBACK
            default: {
                throw std::runtime_error("unrecognized callback");
            }
        }
    }
}
